#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define INTERVALO 1.5
#define THREADS_PER_BLOCK 512
#define NOME_ARQUIVO "resultados_cuda.csv"

__device__ double seno(double x, double y)
{
    return sin(x * x + y * y);
}

__global__ void trapezio_kernel(double hx, double hy, int nx, int ny, float *integral_d)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    float local_integral = 0.0f;
    for (int i = idx; i <= nx; i += total_threads)
    {
        double x = i * hx;
        for (int j = 0; j <= ny; j++)
        {
            double y = j * hy;
            double peso = 1.0;

            if (i == 0 || i == nx)
                peso *= 0.5;
            if (j == 0 || j == ny)
                peso *= 0.5;

            local_integral += peso * seno(x, y);
        }
    }

    atomicAdd(integral_d, local_integral);
}

double trapezio_gpu(int nx, int ny, int num_blocks)
{
    double hx = INTERVALO / nx;
    double hy = INTERVALO / ny;
    float *integral_d;
    float integral_h = 0.0f;

    hipMalloc((void **)&integral_d, sizeof(float));
    hipMemcpy(integral_d, &integral_h, sizeof(float), hipMemcpyHostToDevice);

    trapezio_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(hx, hy, nx, ny, integral_d);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(&integral_h, integral_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(integral_d);

    integral_h *= hx * hy;

    return integral_h;
}

int main()
{
    int intervalos_x[] = {1000, 10000, 100000};
    int intervalos_y[] = {1000, 10000, 100000};
    int blocos[] = {10, 100, 1000};

    FILE *fp = fopen(NOME_ARQUIVO, "w");
    if (!fp)
    {
        printf("Erro ao abrir o arquivo para escrita.\n");
        return 1;
    }
    fprintf(fp, "blocos,intervalo_x,intervalo_y,resultado,tempo\n");

    for (int b = 0; b < 3; b++)
    {
        for (int ix = 0; ix < 3; ix++)
        {
            for (int iy = 0; iy < 3; iy++)
            {
                int nx = intervalos_x[ix];
                int ny = intervalos_y[iy];
                int num_blocks = blocos[b];

                printf("Executando para %d blocos, %d intervalos em x e %d intervalos em y...\n",
                       num_blocks, nx, ny);

                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                hipEventRecord(start, 0);
                double resultado = trapezio_gpu(nx, ny, num_blocks);
                hipEventRecord(stop, 0);

                hipEventSynchronize(stop);
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);

                printf("Resultado: %.10f, Tempo: %.6f segundos\n", resultado, milliseconds / 1000.0);
                fprintf(fp, "%d,%d,%d,%.10f,%.6f\n", num_blocks, nx, ny, resultado, milliseconds / 1000.0);

                hipEventDestroy(start);
                hipEventDestroy(stop);
            }
        }
    }

    fclose(fp);
    return 0;
}
